#include "hip/hip_runtime.h"
#define BLOCK_SIZE 32

/**
 * transpose matrix
 * @param At Transpose of `A`
 * @param A Input matrix
 * @param N number of rows of `A`
 * @param M number of columns of `A`
*/
__global__ void transpose_gpu(double *At, double *A, int N, int M) {
	__shared__ double tile[BLOCK_SIZE][BLOCK_SIZE];

	unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;

	if (y < N && x < M)
        tile[threadIdx.y][threadIdx.x] = A[y*M + x];
	__syncthreads();

	y = blockIdx.x * blockDim.x + threadIdx.y;
	x = blockIdx.y * blockDim.y + threadIdx.x;

	if (y < M && x < N)
        At[y*N + x] = tile[threadIdx.x][threadIdx.y];
}

/**
 * matrix product
 * @param A First matrix
 * @param B Second matrix
 * @param C `A` * `B`
 * @param N number of rows of `A`
 * @param M number of columns of `B`
 * @param P number of columns of `A` and number of rows of `B`
*/
__global__ void product_gpu(double* A, double* B, double* C, int N, int M, int P) {
	uint row = blockIdx.y * blockDim.y + threadIdx.y;
	uint col = blockIdx.x * blockDim.x + threadIdx.x;

	double sum = 0.0;
	__shared__ double As[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ double Bs[BLOCK_SIZE][BLOCK_SIZE];

	uint numBlocks = (P + BLOCK_SIZE - 1) / BLOCK_SIZE;
	for (uint m = 0; m < numBlocks; m++) {
		uint r = m * BLOCK_SIZE + threadIdx.y;
		uint c = m * BLOCK_SIZE + threadIdx.x;
		As[threadIdx.y][threadIdx.x] = A[row*P + c];
		Bs[threadIdx.y][threadIdx.x] = B[r*M + col];
		__syncthreads();

		uint K = BLOCK_SIZE;
        if (m == numBlocks - 1) // last block may be smaller
            K = P - m * BLOCK_SIZE;

		for (uint k = 0; k < K; k++)
			sum += As[threadIdx.y][k] * Bs[k][threadIdx.x];
		__syncthreads();
	}

	if (row < N && col < M)
		C[row * M + col] = sum;
}

/**
 * Extract a submatrix
 * @param B Submatrix of `A`
 * @param A Input matrix
 * @param N rows of `A`
 * @param M columns of `B`
 * @param row_start starting row index (inclusive)
 * @param row_end ending row index (inclusive)
 * @param col_start starting column (inclusive)
 * @param col_end ending column (inclusive)
 */
__global__ void submatrix_gpu(double* A, double* B, int N, int M, int row_start, int row_end, int col_start, int col_end) {
    uint n_cols = col_end - col_start + 1;
    uint row = blockIdx.y * blockDim.y + threadIdx.y;
    uint col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row >= row_start && row <= row_end && col >= col_start && col <= col_end) {
        B[(row-row_start)*n_cols + col - col_start] = A[row*M + col];
    }
}

/**
 * divide all the element of row `c_row` by diagonal element of row `c_row` for both `A` and `I`
 * @param A Input matrix
 * @param I Partial inverse of `A`
 * @param N Number of rows/columns of `A`
 * @param c_row current row
 */
__global__ void inverse_no_diag_division_gpu(double *A, double *I, int N, int c_row){
	int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
        
    __shared__ double diag;

    if(threadIdx.x == 0 && threadIdx.y == 0)
        diag = A[c_row*N + c_row];
    __syncthreads();
    
    if (row == c_row && col != row && row < N && col < N) {
            A[c_row*N + col] = A[c_row*N + col] / diag;
            I[c_row*N + col] = I[c_row*N + col] / diag;
    }
}

/**
 * divide diagonal element of row `c_row` of `I` by diagonal element of row `c_row` of `A`
 * set diagonal element of row `c_row` of A to zero
 * @param A Input matrix
 * @param I Partial inverse of `A`
 * @param N Number of rows/columns of `A`
 * @param c_row current row
 */
__global__ void inverse_diag_division_gpu(double *A, double *I, int N, int c_row){
	int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
        
    if(row == 0 && col == 0) {
        I[c_row*N + c_row] = I[c_row*N + c_row] / A[c_row*N + c_row];
        A[c_row*N + c_row] = 0;
    }
}

/**
 * Gauss-Jordan elimination for matrix inverse
 * @param A Input matrix
 * @param I Partial inverse of `A`
 * @param N Number of rows/columns of `A`
 * @param c_row current row
 */
__global__ void inverse_gauss_jordan_gpu(double *A, double *I, int N, int c_row) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

    __shared__ double tile_A1[BLOCK_SIZE];
    __shared__ double tile_A2[BLOCK_SIZE];
    __shared__ double tile_I1[BLOCK_SIZE];

    if (col < N && row < N) {
        // if I'm the first thread of the row
        if(threadIdx.x == 0) {
            tile_A1[threadIdx.y] = A[row*N + c_row];
        }

        // if I'm the first thread of the column
        if(threadIdx.y == 0) {
            tile_A2[threadIdx.x] = A[c_row*N + col];
            tile_I1[threadIdx.x] = I[c_row*N + col];
        }
    }
    __syncthreads();

	if (row != c_row && col < N && row < N) {        
        I[row*N + col] -= tile_I1[threadIdx.x] * tile_A1[threadIdx.y]; 
        if (col != c_row)
            A[row*N + col] -= tile_A2[threadIdx.x] * tile_A1[threadIdx.y];   
	}
}


/**
 * initialize identity matrix
 * @param A input matrix
 * @param N matrix order
 */
__global__ void inverse_init_identity_gpu(double* A, int N) {
    uint row = blockIdx.y * blockDim.y + threadIdx.y;
    uint col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(row < N && col < N) {
        if (row == col) 
            A[row*N + col] = 1.0;
        else 
            A[row*N + col] = 0.0;
    }
}

/**
 * Moore-Penrose generalized inverse matrix
 * @param G Input matrix
 * @param Y Generalized inverse
 * @param N number of rows of `G` 
 * @param M number of columns of `G`
 * @return execution time
 */
double geninv_gpu(double* G, double* Y, int N, int M) {
    int old_M = M; // to remember M original value
    bool transposed = false; // true if N < M
    dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 grid((M + block.x - 1) / block.x, (N + block.y - 1) / block.y, 1);
    
    //cpu variables
    double* A;
    double* S;

    //gpu variables
    double* d_G;     //original matrix
    double* d_Gt;    // transpose of G
    double* d_A;     // Gt * G
    double* d_S;     // lower triangular of A
    double* d_L;     // lower triangular with zero columns dropped
    double* d_Lt;    // upper triangular with zero rows dropped
    double* d_Lt_L;  // Lt * L
    double* d_I;     // inverse of Lt * L
    double* d_Y;     //pseudoinverse of G

    CHECK( hipMalloc((void**) &d_G, N*M*sizeof(double)) );
    CHECK( hipMalloc((void**) &d_Gt, M*N*sizeof(double)) );
    hipMemcpy(d_G, G, N*M*sizeof(double), hipMemcpyHostToDevice);
    
    double start = seconds();
    
    transpose_gpu<<<grid, block>>>(d_Gt, d_G, N, M); // transpose G in Gt

    // pseudoinverse formula is different if N < M
    if(N < M)  {
        transposed = true;
        M = N;
    }

    hipFree(G);

    A = (double*) malloc(M*M*sizeof(double));
    S = (double*) malloc(M*M*sizeof(double));
    CHECK( hipMalloc((void**) &d_A, M*M*sizeof(double)) );
    CHECK( hipMalloc((void**) &d_S, M*M*sizeof(double)) );
    CHECK( hipMalloc((void**) &d_L, M*M*sizeof(double)) );
    CHECK( hipMalloc((void**) &d_Lt, M*M*sizeof(double)) );
    CHECK( hipMalloc((void**) &d_Lt_L, M*M*sizeof(double)) );
    CHECK( hipMalloc((void**) &d_I, M*M*sizeof(double)) );

    if(transposed)
        product_gpu<<<grid, block>>>(d_G, d_Gt, d_A, N, N, old_M); // A = G * Gt 
    else
        product_gpu<<<grid, block>>>(d_Gt, d_G, d_A, old_M, old_M, N); // A = Gt * G 
    
    hipMemcpy(A, d_A, M*M*sizeof(double), hipMemcpyDeviceToHost);
    int rank = full_rank_cholesky_decomposition(A, S, M); // S = cholesky(A)
    hipMemcpy(d_S, S, M*M*sizeof(double), hipMemcpyHostToDevice);

    submatrix_gpu<<<grid, block>>>(d_S, d_L, M, M, 0, M, 0, rank-1); // S = L with zeros columns dropped
    transpose_gpu<<<grid, block>>>(d_Lt, d_L, M, rank); // transpose of L
    product_gpu<<<grid, block>>>(d_Lt, d_L, d_Lt_L, rank, rank, M); // Lt_L = Lt * L

    // I = inv(Lt_L)
    inverse_init_identity_gpu<<<grid, block>>>(d_I, rank);
    for (int i = 0; i < rank; i++){
        inverse_no_diag_division_gpu <<<grid, block>>>(d_Lt_L, d_I, rank, i);
        inverse_diag_division_gpu <<<1, 1>>>(d_Lt_L, d_I, rank, i);
        inverse_gauss_jordan_gpu <<<grid, block>>>(d_Lt_L, d_I, rank, i);
    }

    double* d_tmp;
    double* d_tmp1; 
    double* d_tmp2;

    if(transposed) { // Y = Gt * L * I * I * Lt
        CHECK( hipMalloc((void**) &d_tmp, old_M*rank*sizeof(double)) );
        CHECK( hipMalloc((void**) &d_tmp1, old_M*rank*sizeof(double)) );
        CHECK( hipMalloc((void**) &d_tmp2, old_M*rank*sizeof(double)) );
        CHECK( hipMalloc((void**) &d_Y, old_M*N*sizeof(double)) );

        product_gpu<<<grid, block>>>(d_Gt, d_L, d_tmp, old_M, rank, N);
        product_gpu<<<grid, block>>>(d_tmp, d_I, d_tmp1, old_M, rank, rank);
        product_gpu<<<grid, block>>>(d_tmp1, d_I, d_tmp2, old_M, rank, rank);
        product_gpu<<<grid, block>>>(d_tmp2, d_Lt, d_Y, old_M, N, rank);
    }
    
    else { // Y = L * I * I * Lt * Gt
        CHECK( hipMalloc((void**) &d_tmp, M*rank*sizeof(double)) );
        CHECK( hipMalloc((void**) &d_tmp1, M*rank*sizeof(double)) );
        CHECK( hipMalloc((void**) &d_tmp2, M*M*sizeof(double)) );
        CHECK( hipMalloc((void**) &d_Y, M*N*sizeof(double)) );

        product_gpu<<<grid, block>>>(d_L, d_I, d_tmp, M, rank, rank);
        product_gpu<<<grid, block>>>(d_tmp, d_I, d_tmp1, M, rank, rank);
        product_gpu<<<grid, block>>>(d_tmp1, d_Lt, d_tmp2, M, M, rank);
        product_gpu<<<grid, block>>>(d_tmp2, d_Gt, d_Y, M, N, M);
    }
    hipDeviceSynchronize();

    double stop = seconds();

    CHECK( hipMemcpy(Y, d_Y, old_M*N*sizeof(double), hipMemcpyDeviceToHost) );

    free(A);
    free(S);
    hipFree(d_Gt);
    hipFree(d_A);
    hipFree(d_I);
    hipFree(d_S);
    hipFree(d_L);
    hipFree(d_Lt);
    hipFree(d_Lt_L);
    hipFree(d_Y);
    hipDeviceReset();

    return stop - start;
}

/**
 * find least squares of a linear system using moore-penrose pseudoinverse
 * @param A Pseudoinverse
 * @param x variables vector
 * @param y constants vector
 * @param N number of equations 
 * @param M number of variables
 */
 void least_square_gpu(double* A, double* x, double* y, int N, int M) {
    dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 grid((M + block.x - 1) / block.x, (N + block.y - 1) / block.y, 1);

    double* d_A;
    double* d_x;
    double* d_y;

    CHECK( hipMalloc((void**) &d_A, M*N*sizeof(double)) );
    CHECK( hipMalloc((void**) &d_x, M*sizeof(double)) );
    CHECK( hipMalloc((void**) &d_y, N*sizeof(double)) );
    CHECK( hipMemcpy(d_A, A, M*N*sizeof(double), hipMemcpyHostToDevice) );
    CHECK( hipMemcpy(d_y, y, N*sizeof(double), hipMemcpyHostToDevice) );
    
    product_gpu<<<grid, block>>>(d_A, d_y, d_x, M, 1, N);
    CHECK( hipMemcpy(x, d_x, M*sizeof(double), hipMemcpyDeviceToHost) );

    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);
    hipDeviceReset();
}